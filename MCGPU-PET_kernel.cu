#include "hip/hip_runtime.h"
//!!MCGPU-PET!!   Kernel called with a 3D grid of blocks with x,y,z sizes equal to the voxel sizes: each block simulates emission from a different voxel.
//!!MCGPU-PET!!   Changes for PET marked with !!MCGPU-PET!!   (ABS, 2017-02-03)


      // *****************************************************************************
      // ***            MCGPU-PET_v0.1  (based on MC-GPU_v1.3)                     ***
      // ***                                                                       ***
      // ***  Distribution:  https://github.com/DIDSR/MCGPU-PET                    ***
      // ***                                                                       ***
      // ***  Authors:                                                             ***
      // ***                                                                       ***
      // ***   MCGPU code foundation and PET source sampling implemented by:       ***
      // ***                                                                       ***
      // ***     - Andreu Badal (Andreu.Badal-Soler[at]fda.hhs.gov)                ***
      // ***          Division of Imaging and Applied Mathematics                  ***
      // ***          Office of Science and Engineering Laboratories               ***
      // ***          Center for Devices and Radiological Health                   ***
      // ***          U.S. Food and Drug Administration                            ***
      // ***                                                                       ***
      // ***                                                                       ***
      // ***   PET detector model and sinogram reporting implemented by:           ***
      // ***                                                                       ***      
      // ***     - Joaquin L. Herraiz and Alejandro López-Montes                   ***  
      // ***         Complutense University of Madrid, EMFTEL, Grupo Fisica Nuclear***
      // ***         and IPARCOS; Instituto de Investigacion Sanitaria Hospital    ***
      // ***         Clinico San Carlos (IdiSSC), Madrid, Spain                    ***
      // ***                                                                       ***
      // ***    Code presented at the IEEE NSS MIC 2021 conference:                ***
      // ***                                                                       ***
      // ***       M-07-01 – GPU-accelerated Monte Carlo-Based Scatter and         ***
      // ***       Prompt-Gamma Corrections in PET, A. López-Montes, J. Cabello,   ***
      // ***       M. Conti, A. Badal, J. L. Herraiz                               ***
      // ***                                                                       ***
      // ***                                                                       ***
      // ***                                      Last update: 2022/02/02          ***
      // *****************************************************************************


////////////////////////////////////////////////////////////////////////////////
//
//              ****************************
//              *** MC-GPU , version 1.3 ***
//              ****************************
//                                          
//!  Definition of the CUDA GPU kernel for the simulation of x ray tracks in a voxelized geometry.
//!  This kernel has been optimized to yield a good performance in the GPU but can still be
//!  compiled in the CPU without problems. All the CUDA especific commands are enclosed in
//!  pre-processor directives that are skipped if the parameter "USING_CUDA" is not defined
//!  at compilation time.
//
//        ** DISCLAIMER **
//
// This software and documentation (the "Software") were developed at the Food and
// Drug Administration (FDA) by employees of the Federal Government in the course
// of their official duties. Pursuant to Title 17, Section 105 of the United States
// Code, this work is not subject to copyright protection and is in the public
// domain. Permission is hereby granted, free of charge, to any person obtaining a
// copy of the Software, to deal in the Software without restriction, including
// without limitation the rights to use, copy, modify, merge, publish, distribute,
// sublicense, or sell copies of the Software or derivatives, and to permit persons
// to whom the Software is furnished to do so. FDA assumes no responsibility
// whatsoever for use by other parties of the Software, its source code,
// documentation or compiled executables, and makes no guarantees, expressed or
// implied, about its quality, reliability, or any other characteristic. Further,
// use of this code in no way implies endorsement by the FDA or confers any
// advantage in regulatory decisions.  Although this software can be redistributed
// and/or modified freely, we ask that any derivative works bear some notice that
// they are derived from it, and any modified versions bear some notice that they
// have been modified.
//                                                                            
//
//!                     @file    MC-GPU_kernel_v1.3.cu
//!                     @author  Andreu Badal (Andreu.Badal-Soler@fda.hhs.gov)
//!                     @date    2012/12/12
//                       -- Original code started on:  2009/04/14
//
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
//!  Initialize the image array, ie, set all pixels to zero
//!  Essentially, this function has the same effect as the command: 
//!   "cutilSafeCall(hipMemcpy(image_device, image, image_bytes, hipMemcpyHostToDevice))";
//!  
//!  CUDA performs some initialization work the first time a GPU kernel is called.
//!  Therefore, calling a short kernel before the real particle tracking is performed
//!  may improve the accuracy of the timing measurements in the relevant kernel.
//!  
//!       @param[in,out] image   Pointer to the image array.
//!       @param[in] pixels_per_image  Number of pixels in the image (ie, elements in the array).
////////////////////////////////////////////////////////////////////////////////
__global__
void init_image_array_GPU(unsigned long long int* image, int pixels_per_image)
{
  int my_pixel = threadIdx.x + blockIdx.x*blockDim.x;
  if (my_pixel < pixels_per_image)
  {
    // -- Set the current pixel to 0 and return, avoiding overflow when more threads than pixels are used:
    image[my_pixel] = (unsigned long long int)(0);    // Initialize non-scatter image
    my_pixel += pixels_per_image;                     //  (advance to next image)
    image[my_pixel] = (unsigned long long int)(0);    // Initialize Compton image
    my_pixel += pixels_per_image;                     //  (advance to next image)
    image[my_pixel] = (unsigned long long int)(0);    // Initialize Rayleigh image
    my_pixel += pixels_per_image;                     //  (advance to next image)
    image[my_pixel] = (unsigned long long int)(0);    // Initialize multi-scatter image
  }
}

// ////////////////////////////////////////////////////////////////////////////////
// //!  Initialize the dose deposition array, ie, set all voxel doses to zero
// //!  
// //!       @param[in,out] dose   Pointer to the dose mean and sigma arrays.
// //!       @param[in] num_voxels_dose  Number of voxels in the dose ROI (ie, elements in the arrays).
// ////////////////////////////////////////////////////////////////////////////////
// __global__
// void init_dose_array_GPU(ulonglong2* voxels_Edep, int num_voxels_dose)
// {  
//   int my_voxel = threadIdx.x + blockIdx.x*blockDim.x;
//   register ulonglong2 ulonglong2_zero;
//   ulonglong2_zero.x = ulonglong2_zero.y = (unsigned long long int) 0;
//   if (my_voxel < num_voxels_dose)
//   {
//     dose[my_voxel] = ulonglong2_zero;    // Set the current voxel to (0,0) and return, avoiding overflow
//   }
// }


 
////////////////////////////////////////////////////////////////////////////////
//!  Main function to simulate x-ray tracks inside a voxelized geometry.
//!  Secondary electrons are not simulated (in photoelectric and Compton 
//!  events the energy is locally deposited).
//!
//!  The following global variables, in  the GPU __constant__ memory are used:
//!           voxel_data_CONST, 
//!           source_energy_data_CONST
//!           mfp_table_data_CONST.
//!
//!       @param[in] history_batch  Particle batch number (only used in the CPU version when CUDA is disabled!, the GPU uses the built-in variable threadIdx)
//!       @param[in] num_p  Projection number in the CT simulation. This variable defines a specific angle and the corresponding source and detector will be used.
//!       @param[in] histories_per_thread   Number of histories to simulate for each call to this function (ie, for GPU thread).
//!       @param[in] seed_input   Random number generator seed (the same seed is used to initialize the two MLCGs of RANECU).
//!       @param[in] voxel_mat_dens   Pointer to the voxel densities and material vector (the voxelized geometry), stored in GPU glbal memory.
//!       @param[in] mfp_Woodcock_table    Two parameter table for the linear interpolation of the Woodcock mean free path (MFP) (stored in GPU global memory).
//!       @param[in] mfp_table_a   First element for the linear interpolation of the interaction mean free paths (stored in GPU global memory).
//!       @param[in] mfp_table_b   Second element for the linear interpolation of the interaction mean free paths (stored in GPU global memory).
//!       @param[in] rayleigh_table   Pointer to the table with the data required by the Rayleigh interaction sampling, stored in GPU global memory.
//!       @param[in] compton_table   Pointer to the table with the data required by the Compton interaction sampling, stored in GPU global memory.
//!       @param[in,out] image   Pointer to the image vector in the GPU global memory.
//!       @param[in,out] dose   Pointer to the array containing the 3D voxel dose (and its uncertainty) in the GPU global memory.
////////////////////////////////////////////////////////////////////////////////
__global__ void track_particles(unsigned long long int* total_histories,
                                int* seed_input_device,
                                PSF_element_struct* PSF,
                                int* index_PSF,
                                ulonglong2* voxels_Edep,
                                float3* voxel_mat_dens,
                                float2* mfp_Woodcock_table,
                                float3* mfp_table_a,
                                float3* mfp_table_b,
                                struct rayleigh_struct* rayleigh_table,
                                struct compton_struct* compton_table,
                                struct detector_struct* detector_data,
                                struct source_struct* source_data,
                                ulonglong2* materials_dose,
			        int* True_dev,
				int* Scatter_dev,
				int* Imagen_T_dev,
				int* Imagen_SC_dev, 
				int* Energy_Spectrum_dev,
                                float E_resol,
                                float E_low,
                                float E_high,
                                float FOVZ,
                                int NROWS,
                                int NCRYSTALS,
                                int NANGLES,
                                int NRAD,
                                int NZS,
                                int NBINS,
                                int RES,
                                int NVOXS,    //FEB2022  !!DeBuG!! Is this input necessary? And should it be NVOX_SIM?
                                int NE,
                                int MRD,
                                int SPAN,
                                int NSINOS)   //JLH (PETA=1-->Trues y Scatter.  PETA=0 -->Bg)
{
    
  // -- Declare the track state variables:
  float3 position, direction, position1, direction1;               // !!MCGPU-PET!! Store sampled values for 1st photon, to use in 2nd
  float energy, step, prob, randno, mfp_density, mfp_Woodcock;
  float3 mfp_table_read_a, mfp_table_read_b;
  int2 seed;
  int index;
  int material0;        // Current material, starting at 0 for 1st material
  int material_old;     // Flag to mark a material or energy change
  signed char scatter_state;    // Flag for scatter images: scatter_state=0 for non-scattered, =1 for Compton, =2 for Rayleigh, and =3 for multiple scatter.
    
  //  Variables for the PSF of the first photon, tallied after a coinicdence with its second is confirmed   !!COINCIDENCE!!
  float energyFirst=-99.9f, travel_distanceFirst=-99.9f;
  float3  positionFirst, directionFirst;
  signed char scatter_stateFirst;

  // -- Store the Compton table in shared memory from global memory:
  //    For Compton and Rayleigh the access to memory is not coherent and the caching capability do not speeds up the accesses, they actually slows down the acces to other data.
  __shared__ struct compton_struct cgco_SHARED;  
  // __shared__ struct detector_struct detector_data_SHARED;
  // __shared__ struct source_struct source_data_SHARED;    
  // Use volatile to prevent storing in registers the shared variable updated by different threads
  volatile __shared__ unsigned long long int acquisition_time_ps_SHARED;    // All threads will generate histories in parallel and increase the shared total acquisition time until total time finished.  !!MCGPU-PET!!
  volatile __shared__ unsigned long long int total_histories_block_SHARED;  // Count all histories generated in this block (ie, this emission voxel)
    //FEB2022  volatile __shared__ double inv_activity_SHARED;    // Never used?                   
  double inv_activity_thread;     // Inverse of the remaining activity in the voxel distributed among threads; gets reduced at each time interval  -- JLH
  __shared__ double inv_mean_life_SHARED;
  __shared__ int tally_TYPE_SHARED;                             // Flag to report only True coincidences (=1), Scatter (=2) or both (=0, default)
  __shared__ int tally_PSF_SINOGRAM_SHARED;                     // Flag to report only PSF (1), SINOGRAM (2) or BOTH (0)  
  
    //FEB2022  int Nvox = gridDim.x*gridDim.y*gridDim.z;  // Number of voxels
    //FEB2022  if (threadIdx.x>=blockDim.x) return;
    //FEB2022  if (blockIdx.x>=gridDim.x || blockIdx.y>=gridDim.y || blockIdx.z>=gridDim.z) return;

  int absvox = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;   // Assuming that the grid size is equal to the voxel geomety size!   !!MCGPU-PET!!    
    //FEB2022  if (absvox<0 || absvox>=Nvox) return;

  // float voxel_activity = source_data->activity[voxel_mat_dens[absvox].x - 1];  //JLH  OLD model: get activity associated to material
  float voxel_activity = voxel_mat_dens[absvox].z;  //JLH   NEW model: get activity for each voxel

  unsigned long long int acquisition_time_ps_thread   = source_data->acquisition_time_ps;           // PET acquisition time in picoseconds (1e-12s) 
  volatile unsigned long long int histories_thread = 0;  

  if (voxel_activity>1.0e-7f) {
    inv_activity_thread = (blockDim.x/(double)voxel_activity);  //JLH  (Voxel Activity distributed among threads);  
      //FEB2022  if (0==threadIdx.x) inv_activity_SHARED = (1.0/(double)voxel_activity);  
  } else {
    return;   // ****** Finish kernel right away if the voxel material does not have any activity (negative input activities also disregarded).
  }

  if (0==threadIdx.x){  // First GPU thread copies the variables to shared memory
   // Copy the compton data to shared memory:
   cgco_SHARED = *compton_table;
   total_histories_block_SHARED = (unsigned long long int)0;
   acquisition_time_ps_SHARED   = source_data->acquisition_time_ps;      // Init PET acquisition time in ps (1e-12s), for all threads !!MCGPU-PET!!    
   inv_mean_life_SHARED         = 1.0/(double)source_data->mean_life; 
   tally_TYPE_SHARED = detector_data->tally_TYPE;       // Keep this constant in shared memory for faster access
   tally_PSF_SINOGRAM_SHARED = detector_data->tally_PSF_SINOGRAM;       // Keep this constant in shared memory for faster access
  }
  __syncthreads();     // Make sure all threads will see the initialized shared variable    

  // -- Initialize the RANECU generator in a position far away from the previous history:  
  int thread_id = (blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y) * blockDim.x + threadIdx.x;  // Using a 1D block and a 3D grid. Assuming grid size = voxel geomety size!  
  int max_histories_per_thread = 555555;      //!!MCGPU-PET!! !!DeBuG!! Assuming that no thread will simulate more than this number of histories!!!
  init_PRNG(thread_id, max_histories_per_thread, *seed_input_device, &seed);  

  //char flag_time_finished = (char)0;  // Flag to identify threads that don't have any more particle to run
  direction1.x = 11.0f;   // Mark that a new photon pair must be sampled   !!MCGPU-PET!!  

  // ---------- Sample and simulate positron annihilation histories until acquisition time finishes: ----------//
  for(;;){  // -------------Infinite Loop to sample Time (MAIN LOOP) --> HISTORIES
    float travel_distance = 0.0f;  // Counter to track the distance traveled by each photon: used to estimate time of flight at detector  !!MCGPU-PET!!
    if (direction1.x>10.0f) {    // We need to sample a new pair of photons, and time will be spent:
    
      //  If (acquisition_time_ps_thread<0.)  break;     // ****Break infinite loop when time has run out for this block! !!MCGPU-PET!!
      //  Sample the time to the next annihilation in picoseconds.     DELTA_T = LOG(1.-RAND(0.0D0)*0.99999)/ACTIV
      //  Each thread in the block will simulate a successive annihilation, and the activity will be reduced accordingly after each event.     // ACTIV   = ACTIV * EXP(-MEAN_LIFE * DELTA_T)
      
      double Dt = -log(ranecu_double(&seed))*inv_activity_thread;  //Time between decays
      
      //for (index=0; index<blockDim.x; index++) {   // Update shared block (=voxel) values secuentially for each thread in the block      
      //  if(threadIdx.x==index) {
      //    if ((unsigned long long int)0==acquisition_time_ps_SHARED)   // Check if a previous thread has finished the time
      //    {         // Time finished
      //      flag_time_finished  = (char)1; 
      //    }
      //    else      // Time remaining
      //    {
      //    Dt = inv_activity_thread * Dt;      
            
      unsigned long long int Dt_ps = __double2ull_rn(Dt*1e12);  // Converted into ps (truncation errors may accumulate --> possible precision loss)
      if (Dt_ps<acquisition_time_ps_thread) {
       acquisition_time_ps_thread = acquisition_time_ps_thread - Dt_ps;       
       histories_thread = histories_thread + 2; //JLH
       //acquisition_time_ps_SHARED   = my_acquisition_time;                  // Update the current acquisition time for all the block, in picoseconds
       //total_histories_block_SHARED = total_histories_block_SHARED + (unsigned long long int)2;     // Update the number of histories simulated (in pairs)
       // JLH
       inv_activity_thread = inv_activity_thread * exp(Dt*inv_mean_life_SHARED);   // Update the value of the activity, one thread at a time    
      } else {       // Time finished           // Sampled time is beyond allowed acquisition time: do not simulate any more
       break;   // ****Break infinite loop when time has run out for this thread! 
       //acquisition_time_ps_SHARED = (unsigned long long int)0;              // Sampled time is beyond allowed acquisition time: do not simulate any more annihilation for this block
       //flag_time_finished  = (char)1;
      } //Check if Time<ACQ_time
    } //direction1.x
 
    //if (flag_time_finished!=0)  continue;   // No particle to track for this thread, skip for loop       !!MCGPU-PET!!  
    //if (absvox==0) printf("**** NEW HISTORY (%d, %d): %lld ps   [seeds: %d, %d]\n",threadIdx.x, thread_id, my_acquisition_time, seed.x, seed.y);   // !!Verbose!!             

    // -- Call the source function to get a primary x ray:      
    source(&energy, &position, &direction, &position1, &direction1, &seed, source_data, &energyFirst); 

    scatter_state = (signed char)0;     // Reset previous scatter state: new non-scattered particle loaded
    // -- Find the current energy bin by truncation:
    index = __float2int_rd((energy-mfp_table_data_CONST.e0)*mfp_table_data_CONST.ide);  // Using CUDA to convert float to integer rounding down       
    // -- Get the minimum mfp at the current energy using linear interpolation (Woodcock tracking):
    float2 mfp_Woodcock_read = mfp_Woodcock_table[index];   // Read the 2 parameters for the linear interpolation in a single read from global memory
    mfp_Woodcock = mfp_Woodcock_read.x + energy * mfp_Woodcock_read.y;   // Interpolated minimum MFP

    // -- Reset previous material to force a recalculation of the MFPs (negative materials are not allowed in the voxels):
    material_old  = -1;    
     
   //break;  //JLH

    // *** X-ray interaction loop:
    for(;;)    {      // MAIN INTERACTION LOOP      
      float3 matdens;
      prob = 0.;     
      absvox = locate_voxel(&position);    // (Returns negative value if particle located outside teh voxelized object bounding box)
      if (absvox<0) break;                 // -- Primary particle was not pointing to the voxel region! (but may still be detected after moving in vacuum in a straight line).
      
      do { // *** Virtual interaction loop:           // New loop structure in MC-GPU_v1.3: simulate all virtual events before sampling Compton & Rayleigh:  
        step = -(mfp_Woodcock)*logf(ranecu(&seed));   // Using the minimum MFP in the geometry for the input energy (Woodcock tracking) 
        travel_distance += step;
        position.x += step*direction.x;
        position.y += step*direction.y;
        position.z += step*direction.z;
        // -- Locate the new particle in the voxel geometry:      
        absvox = locate_voxel(&position);       // Get the voxel number at the current position and the voxel coordinates 
                                                // Used to check if inside the dose ROI in DOSE TALLY.
        if (absvox<0) break;    // Particle escaped the voxel region! ("index" is still >0 at this moment)
        matdens = voxel_mat_dens[absvox];       // Get the voxel material and density in a single read from global memory
        material0 = (int)(matdens.x - 1);       // Set the current material by truncation, and set 1st material to value '0'.
        // -- Get the data for the linear interpolation of the interaction MFPs, in case the energy or material have changed:
        if (material0 != material_old) {
          mfp_table_read_a = mfp_table_a[index*(MAX_MATERIALS)+material0];
          mfp_table_read_b = mfp_table_b[index*(MAX_MATERIALS)+material0];
          material_old = material0;                                              // Store the new material
        }
        // *** Apply Woodcock tracking:
        mfp_density = mfp_Woodcock * matdens.y;
        // -- Calculate probability of delta scattering, using the total mean free path for the current material and energy (linear interpolation):
        prob = 1.0f - mfp_density * (mfp_table_read_a.x + energy * mfp_table_read_b.x);
        randno = ranecu(&seed);    // Sample uniform PRN
      } while (randno<prob);       // [Iterate Do-While Virtual Interaction Loop if there is a delta scattering event]      
      
      if (absvox<0) break;    // -- Particle escaped the voxel region! Break the interaction loop to call tally image.

      // The GPU threads will be stopped and waiting here until ALL threads have a REAL event: 
      // -- Real event takes place! Check the kind of event and sample the effects of the interaction:
      prob += mfp_density * (mfp_table_read_a.y + energy * mfp_table_read_b.y);    // Interpolate total Compton MFP ('y' component)

      if (randno<prob) {  // [Checking Compton scattering]
        // *** Compton interaction:
        //  -- Sample new direction and energy:
        double costh_Compton;
        randno = energy;     // Save temporal copy of the particle energy (variable randno not necessary until next sampling). DOSE TALLY
        GCOa(&energy, &costh_Compton, &material0, &seed, &cgco_SHARED);
        rotate_double(&direction, costh_Compton, 6.28318530717958647693*ranecu_double(&seed));
        randno = energy - randno;   // Save temporal copy of the negative of the energy lost in the interaction.  DOSE TALLY
        // -- Find the new energy interval:
        index = __float2int_rd((energy-mfp_table_data_CONST.e0)*mfp_table_data_CONST.ide);
        if (index>-1) { // 'index' will be negative only when the energy is below the tabulated minimum energy
		        // particle will be then absorbed (rejected) after tallying the dose.
                        // -- Get the Woodcock MFP for the new energy (energy above minimum cutoff):
          float2 mfp_Woodcock_read = mfp_Woodcock_table[index];   // Read the 2 parameters for the linear interpolation in a single read from global memory
          mfp_Woodcock = mfp_Woodcock_read.x + energy * mfp_Woodcock_read.y;   // Interpolated minimum MFP
          material_old = -2;    // Set an impossible material to force an update of the MFPs data for the nex energy interval
          // -- Update scatter state:
          if (scatter_state==(signed char)0){
            scatter_state = (signed char)1;   // Set scatter_state == 1: Compton scattered particle
          }else{
            scatter_state = (signed char)3;   // Set scatter_state == 3: Multi-scattered particle
          }
        } //Index
      } else {
         prob += mfp_density * (mfp_table_read_a.z + energy * mfp_table_read_b.z);    // Interpolate total Rayleigh MFP ('z' component)
         if (randno<prob) {   // [Checking Rayleigh scattering]
          // *** Rayleigh interaction: -- Sample angular deflection:
          double costh_Rayleigh;
          float pmax_current = rayleigh_table->pmax[(index+1)*MAX_MATERIALS+material0];   // Get max (ie, value for next bin?) 
                                                                                          //cumul prob square form factor for Rayleigh sampling
          GRAa(&energy, &costh_Rayleigh, &material0, &pmax_current, &seed, rayleigh_table);
          rotate_double(&direction, costh_Rayleigh, 6.28318530717958647693*ranecu_double(&seed));
          // -- Update scatter state:
          if (scatter_state==(signed char)0) {
             scatter_state = (signed char)2;   // Set scatter_state == 1: Rayleigh scattered particle
          } else {
            scatter_state = (signed char)3;   // Set scatter_state == 3: Multi-scattered particle
          }
         } else {
           // *** Photoelectric interaction (or pair production): mark particle for absorption after dose tally (ie, index<0)!
           randno = -energy;   // Save temporal copy of the (negative) energy deposited in the interaction (variable randno not necessary anymore).
           index = -11;       // A negative "index" marks that the particle was absorbed and that it will never arrive at the detector.
         } //randno vs prob
      } // index
    
      //  -- Tally the dose deposited in Compton and photoelectric interactions:
      if (randno<-0.001f) {
        float Edep = -1.0f*randno;   // If any energy was deposited, this variable will temporarily store the negative value of Edep.
        //  -- Tally the dose deposited in the current material, if enabled (ie, array allocated and not null):
        if (materials_dose!=NULL) tally_materials_dose(&Edep, &material0, materials_dose);    // !!tally_materials_dose!!
        //  -- Tally the energy deposited in the current voxel, if enabled (tally disabled when dose_ROI_x_max_CONST is negative). DOSE TALLY
        if (dose_ROI_x_max_CONST > -1) {
          short3 voxel_coord;
          voxel_coord.x = __float2int_rd(position.x * voxel_data_CONST.inv_voxel_size.x);   // !!MCGPU-PET!!  
          voxel_coord.y = __float2int_rd(position.y * voxel_data_CONST.inv_voxel_size.y);
          voxel_coord.z = __float2int_rd(position.z * voxel_data_CONST.inv_voxel_size.z);
          // CODE TO MAP EMISSION VOXELS INSTEAD OF REAL DOSE DEPOSITION LOCATIONS: v
          //voxel_coord.x=blockIdx.x; voxel_coord.y=blockIdx.y; voxel_coord.z=blockIdx.z;  
          tally_voxel_energy_deposition(&Edep, &voxel_coord, voxels_Edep);
        }
      }    

      // -- Break interaction loop for particles absorbed or with energy below the tabulated cutoff: particle is "absorbed" (ie, track discontinued).
      if (index<0) break;  
      
    }   // [Cycle the X-ray interaction loop]
    
    //!!COINCIDENCE!!
    if (index>-1) {
      // Particle escaped geometry and not absorbed: chance of coincidence detection
      if (direction1.x>10.0f) {    // Simulating the second particle of a pair (set by source routine).      !!COINCIDENCE!!
        if (energyFirst>-0.5f) {    // True if the first particle was not absorbed
	  // Both pair particle escaped the voxels but were not absorbed, check if it will arrive at the detector and tally PSF:
          // Report only PSF (tally_PSF_SINOGRAM==1), SINOGRAM (tally_PSF_SINOGRAM==2) or both (tally_PSF_SINOGRAM==0)
          // Report only True coincidences (tally_TYPE==1), Scatter (tally_TYPE==2) or both (tally_TYPE==0)
          if (tally_TYPE_SHARED==0 || (tally_TYPE_SHARED==1 && scatter_state==0 && scatter_stateFirst==0) || (tally_TYPE_SHARED==2 && (scatter_state!=0 || scatter_stateFirst!=0)))  {  
          if ((tally_PSF_SINOGRAM_SHARED==0)||(tally_PSF_SINOGRAM_SHARED==2))  {
            tally_Sinogram(&energy, &energyFirst, &position, &positionFirst, &direction, &directionFirst, &scatter_state, &scatter_stateFirst, detector_data, source_data, &acquisition_time_ps_thread, &travel_distance, &travel_distanceFirst, True_dev, Scatter_dev, Imagen_T_dev, Imagen_SC_dev, Energy_Spectrum_dev, &seed, &E_resol, &E_low, &E_high, &FOVZ, &NROWS, &NCRYSTALS, &NANGLES, &NRAD, &NZS, &NBINS, &RES, &NVOXS, &NE, &MRD, &SPAN, &NSINOS);
          }
          if ((tally_PSF_SINOGRAM_SHARED==0)||(tally_PSF_SINOGRAM_SHARED==1))  { 
           tally_PSF_coincidences(&energy, &energyFirst, &position, &positionFirst, &direction, &directionFirst, &scatter_state, &scatter_stateFirst, detector_data, source_data, &acquisition_time_ps_thread, &travel_distance, &travel_distanceFirst, PSF, index_PSF, &seed, &E_resol, &E_low, &E_high);    // !!MCGPU-PET!!    // !!COINCIDENCE!! 
          } // PSF/Sinogram output
         } // True/Scatter type
        } //Energyfirst
      } else { 
       // Store state of first particle of the coinicidence event for reporting later to PSF     // !!COINCIDENCE!!
       energyFirst = energy;
       positionFirst = position;
       directionFirst = direction;
       scatter_stateFirst = scatter_state;
       travel_distanceFirst = travel_distance;
      } //direction
    } // Index


// OLD CODE NOT REPORTING COINCIDENCES ONLY:
// -- Particle escaped the voxels but was not absorbed, check if it will arrive at the detector and tally its energy:
//       tally_PSF(&energy, &position, &direction, &scatter_state, detector_data, source_data, &my_acquisition_time, &travel_distance, PSF, index_PSF);      // !!MCGPU-PET!! 

  }   // [Continue with a new history]
  
    
  // -- Store the final random seed used by the last thread in the grid to global memory in order to continue the random secuence in successive kernel executions in the same GPU.
  //    Since I am only storing the 'x' component and using it to init both parts of the ranecu generator, the secuence will actually diverge, but I warranty that at least one MLCG will stay uncorrelated.
  if ( (blockIdx.x == (gridDim.x-1)) && (blockIdx.y == (gridDim.y-1)) && (blockIdx.z == (gridDim.z-1)) && (threadIdx.x == (blockDim.x-1)))  
  { *seed_input_device = seed.x; }   // Store last seed used in last kernel thread

  atomicAdd(total_histories,histories_thread);     
  __syncthreads();     // Make sure all threads are done simulating histories

  //if (0==threadIdx.x) {atomicAdd(total_histories, total_histories_block_SHARED);   }
  // Safely add the histories generated by all threads in this block to the complete grid total in global memory (only 1 thread per block will call atomicAdd)
 
}   // [All tracks simulated for this kernel call: return to CPU]



////////////////////////////////////////////////////////////////////////////////
//!  Tally the dose deposited in the voxels.
//!  This function is called whenever a particle suffers a Compton or photoelectric
//!  interaction. It is not necessary to call this function if the dose tally
//!  was disabled in the input file (ie, dose_ROI_x_max_CONST < 0).
//!  Electrons are not transported in MC-GPU and therefore we are approximating
//!  that the dose is equal to the KERMA (energy released by the photons alone).
//!  This approximation is acceptable when there is electronic equilibrium and when
//!  the range of the secondary electrons is shorter than the voxel size. Usually the
//!  doses will be acceptable for photon energies below 1 MeV. The dose estimates may
//!  not be accurate at the interface of low density volumes.
//!
//!  We need to use atomicAdd() in the GPU to prevent that multiple threads update the 
//!  same voxel at the same time, which would result in a lose of information.
//!  This is very improbable when using a large number of voxels but gives troubles 
//!  with a simple geometries with few voxels (in this case the atomicAdd will slow 
//!  down the code because threads will update the voxel dose secuentially).
//!
//!
//!       @param[in] Edep   Energy deposited in the interaction
//!       @param[in] voxel_coord   Voxel coordinates, needed to check if particle located inside the input region of interest (ROI)
//!       @param[out] voxels_Edep   ulonglong2 array containing the 3D voxel dose and dose^2 (ie, uncertainty) as unsigned integers scaled by SCALE_eV.
////////////////////////////////////////////////////////////////////////////////
__device__  void tally_voxel_energy_deposition(float* Edep, short3* voxel_coord, ulonglong2* voxels_Edep) {

    // !!DeBuG!! Maybe it would be faster to store a 6 element struct and save temp copy?? struct_short_int_x6_align16  dose_ROI_size = dose_ROI_size_CONST;   // Get ROI coordinates from GPU constant memory and store temporal copy
  
  if((voxel_coord->x < dose_ROI_x_min_CONST) || (voxel_coord->x > dose_ROI_x_max_CONST) ||
     (voxel_coord->y < dose_ROI_y_min_CONST) || (voxel_coord->y > dose_ROI_y_max_CONST) ||
     (voxel_coord->z < dose_ROI_z_min_CONST) || (voxel_coord->z > dose_ROI_z_max_CONST))
    {
      return;   // -- Particle outside the ROI: return without tallying anything.
    }

  // -- Particle inside the ROI: tally Edep.
  register int DX = 1 + (int)(dose_ROI_x_max_CONST - dose_ROI_x_min_CONST);
  register int num_voxel = (int)(voxel_coord->x-dose_ROI_x_min_CONST) + ((int)(voxel_coord->y-dose_ROI_y_min_CONST))*DX + ((int)(voxel_coord->z-dose_ROI_z_min_CONST))*DX*(1 + (int)(dose_ROI_y_max_CONST-dose_ROI_y_min_CONST));
  
   #ifdef USING_CUDA
     atomicAdd(&voxels_Edep[num_voxel].x, __float2ull_rn((*Edep)*SCALE_eV) );    // Energy deposited at the voxel, scaled by the factor SCALE_eV and rounded.
     atomicAdd(&voxels_Edep[num_voxel].y, __float2ull_rn((*Edep)*(*Edep)) );     // (not using SCALE_eV for std_dev to prevent overflow)           
   #else
     voxels_Edep[num_voxel].x += (unsigned long long int)((*Edep)*SCALE_eV + 0.5f);
     voxels_Edep[num_voxel].y += (unsigned long long int)((*Edep)*(*Edep) + 0.5f);
   #endif
          
  return;
}

//   // !!COINCIDENCE!!     [December 13, 2017]
////////////////////////////////////////////////////////////////////////////////
//!  Tally a Phase Space File (PSF) of the COINCIDENCE photons escaping the voxelized volume.
//!  If one of the photons is absorbed, none is reported in PSF. If no absorption happens, both 
//!  particles are reported in consecutive PSF elements, sorted by shortest arrival time.
//!
//!       @param[in] energy    X-ray energy
//!       @param[in] position  Particle position
//!       @param[in] direction Particle direction (cosine vectors)
//!       @param[in] scatter_state  Flag marking primaries, single Compton, single Rayleigh or multiple scattered radiation
//!       @param[in] detector_data  Variables defining the detector geometry, stored in shared memory for fast access.
//!       @param[out] image    Pointer to the global memory array storing the PSF
////////////////////////////////////////////////////////////////////////////////

__device__
inline void tally_PSF_coincidences(float* energy, float* energyFirst, float3* position, float3* positionFirst, float3* direction, float3* directionFirst, signed char* scatter_state, signed char* scatter_stateFirst, struct detector_struct* detector_data, struct source_struct* source_data, unsigned long long int* acquisition_time_ps_thread, float* travel_distance, float* travel_distanceFirst, PSF_element_struct* PSF, int* index_PSF, int2 *seed, float* E_resol, float* E_low, float* E_high)     // !!MCGPU-PET!!    // !!COINCIDENCE!!
{
  //! Assuming a cylindrical detector with central axis in the Z direction. The cylinder must be larger than the voxel volume (negative distances not computed)       // !!MCGPU-PET!!
   
  // -- Move photon to the surface of the cylinder (radius in cylindrical coordinates equals PSF_radius):
  float x0 = position->x - detector_data->PSF_center.x;   // Move to coordinate system with cylinder center at origin
  float y0 = position->y - detector_data->PSF_center.y;
  
  float A = direction->x*direction->x + direction->y*direction->y;
  float B = 2.0f*(x0*direction->x + y0*direction->y);
  float C = x0*x0 + y0*y0 - detector_data->PSF_radius*detector_data->PSF_radius;
    
  float dist;
  if (A==0.0f) {
    dist = 99999999.9f;   // Photon moving along Z axis -> it will intersect the cylinder only at infinite distance
  } else {
    dist = (-B + sqrtf(B*B-4.0f*A*C)) / (2.0f*A);    // There will always be a real, positive dist as long as the cylinder is larger than the voxels (no particle is ever outside the cylinder)
  }  
  
  *travel_distance += dist;
  position->x = x0 + dist*direction->x;
  position->y = y0 + dist*direction->y;
  position->z = (position->z - detector_data->PSF_center.z) + dist*direction->z;
  float phi = atan2f(position->y, position->x);
  
  // -- Report only particles that intersect the cylinder inside the PSF detector height:
  if (fabsf(position->z)<0.5f*detector_data->PSF_height) {    
    
    // Repeat same calculations for second photon:
    x0 = positionFirst->x - detector_data->PSF_center.x;   // Move to coordinate system with cylinder center at origin
    y0 = positionFirst->y - detector_data->PSF_center.y;
  
    A = directionFirst->x*directionFirst->x + directionFirst->y*directionFirst->y;
    B = 2.0f*(x0*directionFirst->x + y0*directionFirst->y);
    C = x0*x0 + y0*y0 - detector_data->PSF_radius*detector_data->PSF_radius;
    
    if (A==0.0f) {
      dist = 99999999.9f;   // Photon moving along Z axis -> it will intersect the cylinder only at infinite distance
    } else {
      dist = (-B + sqrtf(B*B-4.0f*A*C)) / (2.0f*A);    // There will always be a real, positive dist as long as the cylinder is larger than the voxels (no particle is ever outside the cylinder)
    }  

    *travel_distanceFirst += dist;
    positionFirst->x = x0 + dist*directionFirst->x;
    positionFirst->y = y0 + dist*directionFirst->y;
    positionFirst->z = (positionFirst->z - detector_data->PSF_center.z) + dist*directionFirst->z;
    float phi1 = atan2f(positionFirst->y, positionFirst->x);
    
  // Energy response of the detector (energy resolution)     // JLH
 	float randno1 = ranecu(seed);   
	float randno2 = ranecu(seed);   
	float gaussian_var1 = sqrtf(-2.0f*logf(randno1+1e-8f))*cosf(2.0f*PI*randno2);
	float randno3 = ranecu(seed);   
	float randno4 = ranecu(seed);   
	float gaussian_var2 = sqrtf(-2.0f*logf(randno3+1.0e-8f))*cosf(2.0f*PI*randno4);
	float Energia1 = (*energy) + ((*E_resol)*(1.0f/2.35f))*(*energy)*gaussian_var1;
	float Energia2 = (*energyFirst) + ((*E_resol)*(1.0f/2.35f))*(*energyFirst)*gaussian_var2;

    // -- Report only particles that intersect the cylinder inside the PSF detector height:
    if (fabsf(positionFirst->z)<0.5f*detector_data->PSF_height && (Energia1 > *E_low) && (Energia1 < *E_high) && (Energia2 > *E_low) && (Energia2 < *E_high)){
  
      // Safely get two slots in the PSF array in global memory:
      int index = atomicAdd(index_PSF, 2);
      
      if (index>2000000000)
        *index_PSF = 2000000000;    // Prevent overflow of integer counter (max value 2^31-1~2.14e9)      //!!DeBuG!!
      
      if (index<detector_data->PSF_size) { // Avoid overflow!
        
        // Store particles in order of arrival (shortest travel first):
        int i0, i1;
        if (*travel_distance<*travel_distanceFirst)
        {
          i0=index;
          i1=index+1;
        }
        else
        {
          i0=index+1;
          i1=index;
        }
        
        
       // Store first particle data in global memory:
        PSF[i0].emission_time_ps = source_data->acquisition_time_ps - *acquisition_time_ps_thread;             // Report emission time starting from time 0 s
        PSF[i0].travel_time_ps   = (*travel_distance)*inv_SPEEDOFLIGHT;                                 // Convert the distance to picoseconds (assuming speed of light in vacuum for every material)
        PSF[i0].emission_absvox  = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;  // Assuming that the grid size is equal to the voxel geomety size!   !!MCGPU-PET!!  
        PSF[i0].energy = Energia1;
        PSF[i0].z      = position->z;
        PSF[i0].phi    = phi;
        PSF[i0].vx     = direction->x;
        PSF[i0].vy     = direction->y;
        PSF[i0].vz     = direction->z;
        PSF[i0].index1 = (short int)(*scatter_state);   // Flag for scatter: =0 for non-scattered, =1 for Compton, =2 for Rayleigh, and =3 for multiple scatter.
        PSF[i0].index2 = (short int)0;                  // use not defined yet (decay, prompt...)
        
        // Store second particle data in global memory:
        PSF[i1].emission_time_ps = source_data->acquisition_time_ps - *acquisition_time_ps_thread;             // Report emission time starting from time 0 s
        PSF[i1].travel_time_ps   = (*travel_distanceFirst)*inv_SPEEDOFLIGHT;                                 // Convert the distance to picoseconds (assuming speed of light in vacuum for every material)
        PSF[i1].emission_absvox  = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;  // Assuming that the grid size is equal to the voxel geomety size!   !!MCGPU-PET!!  
        PSF[i1].energy = Energia2;
        PSF[i1].z      = positionFirst->z;
        PSF[i1].phi    = phi1;
        PSF[i1].vx     = directionFirst->x;
        PSF[i1].vy     = directionFirst->y;
        PSF[i1].vz     = directionFirst->z;
        PSF[i1].index1 = (short int)(*scatter_stateFirst);   // Flag for scatter: =0 for non-scattered, =1 for Compton, =2 for Rayleigh, and =3 for multiple scatter.
        PSF[i1].index2 = (short int)0;                  // use not defined yet (decay, prompt...)
      }
      
    }
  }

}





////////////////////////////////////////////////////////////////////////////////
//!  Source that creates primary x rays, according to the defined source model.
//!  The particles are automatically moved to the surface of the voxel bounding box,
//!  to start the tracking inside a real material. If the sampled particle do not
//!  enter the voxels, it is init in the focal spot and the main program will check
//!  if it arrives at the detector or not.
//!
//!       @param[in] source_data   Structure describing the source.
//!       @param[in] source_energy_data_CONST   Global variable in constant memory space describing the source energy spectrum.
//!       @param[out] position   Initial particle position (particle transported inside the voxel bbox).
//!       @param[out] direction   Sampled particle direction (cosine vectors).
//!       @param[out] energy   Sampled energy of the new x ray.
//!       @param[in] seed   Current seed of the random number generator, requiered to sample the movement direction.
//!       @param[out] absvox   Set to <0 if primary particle will not cross the voxels, not changed otherwise (>0).
////////////////////////////////////////////////////////////////////////////////
__device__
inline void source(float* energy, float3* position, float3* direction, float3* position1, float3* direction1, int2* seed, struct source_struct* source_data, float* energyFirst)
{
  // *** Assign annihilation photon energy:
  *energy = ANNIHILATION_PHOTON_ENERGY;        // Set initial energy of 510998.95 eV



  if (direction1->x>10.0f)    // First photon of the pair: we need to sample
  {  
    // *** Sample the initial direction isotropically:   
    direction->z = 1.0f - 2.0f*ranecu(seed); //--cosine of theta (spherical coordinates phi,theta)
    register float phi_sampled = 6.28318530716f*ranecu(seed);     // Angle in (0, 2*pi)
    register float theta_sampled = acosf(direction->z);
    register float sin_theta_sampled = sqrtf(1.0f - direction->z*direction->z); //--sine of theta
    float sinphi_sampled, cosphi_sampled;
    sincosf(phi_sampled, &sinphi_sampled,&cosphi_sampled);    // Calculate the SIN and COS at the same time.    
    direction->y = sin_theta_sampled * sinphi_sampled; //spherical coordinates
    direction->x = sin_theta_sampled * cosphi_sampled; //spherical coordinates

    // *** Sample x ray emission position uniformly inside the voxel:
    //     Making sure that the particle is not exactly on the surface of the voxel to prevent floating point errors to cause the emission from the wrong absvox:
    float r1 = ranecu(seed), r2 = ranecu(seed), r3 = ranecu(seed);
    r1 = max_value(EPS_SOURCE, r1-EPS_SOURCE), r2 = max_value(EPS_SOURCE, r2-EPS_SOURCE), r3 = max_value(EPS_SOURCE, r3-EPS_SOURCE); 
  
    position->x = ((float)blockIdx.x + r1)/voxel_data_CONST.inv_voxel_size.x;   // Assuming that grid size equal to voxel geomety size!  !!MCGPU-PET!!
    position->y = ((float)blockIdx.y + r2)/voxel_data_CONST.inv_voxel_size.y;
    position->z = ((float)blockIdx.z + r3)/voxel_data_CONST.inv_voxel_size.z;

    
    direction1->x = direction->x;   // Store sampled position and direction
    direction1->y = direction->y;
    direction1->z = direction->z;
    position1->x = position->x;
    position1->y = position->y;
    position1->z = position->z;
    
    *energyFirst = -10.0f;   // By default, mark that first particle in pair was absorbed -> state changed later if absorption does not happen.     !!COINCIDENCE!!
    
    return;    // Exit function
  }
  
  //else                       // Second photon of the pair: 
			       // JLH // Instead of using previously sampled info, we generate it again
  //{
     
    if (PETA_DEV[0]==0) {
    //JLH--- random emission of photon 2 ---for bg simulation
     direction->z = 1.0f - 2.0f*ranecu(seed);
     register float phi_sampled = 6.28318530716f*ranecu(seed);     // Angle in (0, 2*pi)
     register float sin_theta_sampled = sqrtf(1.0f - direction->z*direction->z);
     float sinphi_sampled, cosphi_sampled;
     sincosf(phi_sampled, &sinphi_sampled,&cosphi_sampled);    // Calculate the SIN and COS at the same time.    
     direction->y = sin_theta_sampled * sinphi_sampled;
     direction->x = sin_theta_sampled * cosphi_sampled;
    }
    else{  //180 deg emission // Standard PET
     //---------NO-COLLINEARITY--------------
     register float phi_sampled2 = atan2f(direction1->y,direction1->x)+PI; //collinearity
     register float theta_sampled2 = PI - acos(direction1->z); //collinearity
            
      float V1 = ranecu(seed);
      //float V2 = ranecu(seed)*2-1;
      float V2 = ranecu(seed);
      //float V_2 = V1*V1+V2*V2;
      //float THETA_NC = sqrtf(-2*SIGMA_NC*SIGMA_NC*2*logf(V2*V2)/V_2)*V1; 
      float THETA_NC = (2.0*0.21276*PI/180.0)*sqrtf(-2.0f*logf(V2+1e-8f))*cosf((2.0*PI)*V1);    // float SIGMA_NC = 2.0*0.21276*PI/180.0;
      float PHI_NC = (2.0*PI)*ranecu(seed);
      
      float ST,CT,CP,SP, ST1,CT1,CP1,SP1;
      sincosf(THETA_NC,      &ST, &CT);    // Calculate the SIN and COS at the same time.    
      sincosf(PHI_NC,        &SP, &CP); 
      sincosf(theta_sampled2,&ST1,&CT1);
      sincosf(phi_sampled2,  &SP1,&CP1);
          //FEB2022   float ST = sinf(THETA_NC), CT = cosf(THETA_NC), CP = cosf(PHI_NC), SP = sinf(PHI_NC), ST1 = sinf(theta_sampled2), CT1 = cosf(theta_sampled2), SP1 = sinf(phi_sampled2), CP1 = cosf(phi_sampled2);

      float GX = CP*ST;
      float GY = SP*ST;
      float GZ = CT;

      float GX1 = GX*CT1*CP1-GY*SP1+GZ*ST1*CP1;
      float GY1 = GX*CT1*SP1+GY*CP1+GZ*ST1*SP1;
      float GZ1 = -GX*ST1+GZ*CT1;   

      direction->x = GX1;
      direction->y = GY1;
      direction->z = GZ1;

      //theta_sampled2 = acos(GZ1);
      //phi_sampled2 = atan2f(GY1,GX1);

      //direction->y = sin(theta_sampled2) * sin(phi_sampled2);
      //direction->x = sin(theta_sampled2) * cos(phi_sampled2);
      //direction->z = cos(theta_sampled2);
   
     //----END OF NO-COLLINEARITY-----------
    
     //direction->x = -direction1->x;
     //direction->y = -direction1->y;
     //direction->z = -direction1->z;
    // --- end of random JLH ------
    }

    position->x = position1->x;
    position->y = position1->y;
    position->z = position1->z;    
    direction1->x = 11.1f;   // Mark that a new photon must be sampled next time
//   }
}



////////////////////////////////////////////////////////////////////////////////
//!  Functions that moves a particle inside the voxelized geometry bounding box.
//!  An EPSILON distance is added to make sure the particles will be clearly inside the bbox, 
//!  not exactly on the surface. 
//!
//!  This algorithm makes the following assumtions:
//!     - The back lower vertex of the voxel bounding box is always located at the origin: (x0,y0,z0)=(0,0,0).
//!     - The initial value of "position" corresponds to the focal spot location.
//!     - When a ray is not pointing towards the bbox plane that it should cross according to the sign of the direction,
//!       I assign a distance to the intersection =0 instead of the real negative distance. The wall that will be 
//!       crossed to enter the bbox is always the furthest and therefore a 0 distance will never be used except
//!       in the case of a ray starting inside the bbox or outside the bbox and not pointing to any of the 3 planes. 
//!       In this situation the ray will be transported a 0 distance, meaning that it will stay at the focal spot.
//!
//!  (Interesting information on ray-box intersection: http://tog.acm.org/resources/GraphicsGems/gems/RayBox.c)
//!
//!       @param[in,out] position Particle position: initially set to the focal spot, returned transported inside the voxel bbox.
//!       @param[out] direction   Sampled particle direction (cosine vectors).
//!       @param[out] intersection_flag   Set to <0 if particle outside bbox and will not cross the voxels, not changed otherwise.
//!       @param[out] size_bbox   Size of the bounding box.
////////////////////////////////////////////////////////////////////////////////
/*   [FUNCTION NOT USED IN MCGPU-PET BECAUSE THE PHOTONS START INSIDE THE VOXELIZED GEOMETRY BY DESIGN]
__device__
inline void move_to_bbox(float3* position, float3* direction, float3 size_bbox, int* intersection_flag)
{
  float dist_y, dist_x, dist_z;

  // -Distance to the nearest Y plane:
  if ((direction->y) > EPS_SOURCE)   // Moving to +Y: check distance to y=0 plane
  {
    // Check Y=0 (bbox wall):
    if (position->y > 0.0f)  // The input position must correspond to the focal spot => position->y == source_data_CONST.position[*num_p].y
      dist_y = 0.0f;  // No intersection with this plane: particle inside or past the box  
          // The actual distance would be negative but we set it to 0 bc we will not move the particle if no intersection exist.
    else
      dist_y = EPS_SOURCE + (-position->y)/(direction->y);    // dist_y > 0 for sure in this case
  }
  else if ((direction->y) < NEG_EPS_SOURCE)
  {
    // Check Y=voxel_data_CONST.size_bbox.y:
    if (position->y < size_bbox.y)
      dist_y = 0.0f;  // No intersection with this plane
    else
      dist_y = EPS_SOURCE + (size_bbox.y - position->y)/(direction->y);    // dist_y > 0 for sure in this case
  }
  else   // (direction->y)~0
    dist_y = NEG_INF;   // Particle moving parallel to the plane: no interaction possible (set impossible negative dist = -INFINITE)

  // -Distance to the nearest X plane:
  if ((direction->x) > EPS_SOURCE)
  {
    // Check X=0:
    if (position->x > 0.0f)
      dist_x = 0.0f;
    else  
      dist_x = EPS_SOURCE + (-position->x)/(direction->x);    // dist_x > 0 for sure in this case
  }
  else if ((direction->x) < NEG_EPS_SOURCE)
  {
    // Check X=voxel_data_CONST.size_bbox.x:
    if (position->x < size_bbox.x)
      dist_x = 0.0f;
    else  
      dist_x = EPS_SOURCE + (size_bbox.x - position->x)/(direction->x);    // dist_x > 0 for sure in this case
  }
  else
    dist_x = NEG_INF;

  // -Distance to the nearest Z plane:
  if ((direction->z) > EPS_SOURCE)
  {
    // Check Z=0:
    if (position->z > 0.0f)
      dist_z = 0.0f;
    else
      dist_z = EPS_SOURCE + (-position->z)/(direction->z);    // dist_z > 0 for sure in this case
  }
  else if ((direction->z) < NEG_EPS_SOURCE)
  {
    // Check Z=voxel_data_CONST.size_bbox.z:
    if (position->z < size_bbox.z)
      dist_z = 0.0f;
    else
      dist_z = EPS_SOURCE + (size_bbox.z - position->z)/(direction->z);    // dist_z > 0 for sure in this case
  }
  else
    dist_z = NEG_INF;

  
  // -- Find the longest distance plane, which is the one that has to be crossed to enter the bbox.
  //    Storing the maximum distance in variable "dist_z". Distance will be =0 if no intersection exists or 
  //    if the x ray is already inside the bbox.
  if ( (dist_y>dist_x) && (dist_y>dist_z) )
    dist_z = dist_y;      // dist_z == dist_max 
  else if (dist_x>dist_z)
    dist_z = dist_x;
// else
//   dist_max = dist_z;
    
  // -- Move particle from the focal spot (current location) to the bbox wall surface (slightly inside):
  position->x += dist_z * direction->x;
  position->y += dist_z * direction->y;
  position->z += dist_z * direction->z;      
  
  // Check if the new position is outside the bbox. If true, the particle must be moved back to the focal spot location:
  if ( (position->x < 0.0f) || (position->x > size_bbox.x) || 
       (position->y < 0.0f) || (position->y > size_bbox.y) || 
       (position->z < 0.0f) || (position->z > size_bbox.z) )
  {
    position->x -= dist_z * direction->x;  // Reject new position undoing the previous translation
    position->y -= dist_z * direction->y;
    position->z -= dist_z * direction->z;
    (*intersection_flag) = -111;  // Particle outside the bbox AND not pointing to the bbox: set absvox<0 to skip interaction sampling.
  }
}
*/

////////////////////////////////////////////////////////////////////////////////


//!  Upper limit of the number of random values sampled in a single track.
#define  LEAP_DISTANCE     256
//!  Multipliers and moduli for the two MLCG in RANECU.
#define  a1_RANECU       40014
#define  m1_RANECU  2147483563
#define  a2_RANECU       40692
#define  m2_RANECU  2147483399
////////////////////////////////////////////////////////////////////////////////
//! Initialize the pseudo-random number generator (PRNG) RANECU to a position
//! far away from the previous history (leap frog technique).
//!
//! Each calculated seed initiates a consecutive and disjoint sequence of
//! pseudo-random numbers with length LEAP_DISTANCE, that can be used to
//! in a parallel simulation (Sequence Splitting parallelization method).
//! The basic equation behind the algorithm is:
//!    S(i+j) = (a**j * S(i)) MOD m = [(a**j MOD m)*S(i)] MOD m  ,
//! which is described in:
//!   P L'Ecuyer, Commun. ACM 31 (1988) p.742
//!
//! This function has been adapted from "seedsMLCG.f", see:
//!   A Badal and J Sempau, Computer Physics Communications 175 (2006) p. 440-450
//!
//!       @param[in] history   Particle bach number.
//!       @param[in] seed_input_device   Initial PRNG seed input (used to initiate both MLCGs in RANECU).
//!       @param[out] seed   Initial PRNG seeds for the present history.
//!
////////////////////////////////////////////////////////////////////////////////
__device__
inline void init_PRNG(int history_batch, int histories_per_thread, int seed_input, int2* seed)
{
  // -- Move the RANECU generator to a unique position for the current batch of histories:
  //    I have to use an "unsigned long long int" value to represent all the simulated histories in all previous batches
  //    The maximum unsigned long long int value is ~1.8e19: if history >1.8e16 and LEAP_DISTANCE==1000, 'leap' will overflow.
  // **** 1st MLCG:
  unsigned long long int leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
  int y = 1;
  int z = a1_RANECU;
  // -- Calculate the modulo power '(a^leap)MOD(m)' using a divide-and-conquer algorithm adapted to modulo arithmetic
  for(;;)
  {
    // (A2) Halve n, and store the integer part and the residue
    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
    {
      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);  
      y = abMODm(m1_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
      if (0==leap) break;         // (A4) leap==0? ==> finish
    }
    else           // (leap is even)
    {
      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
    }
    z = abMODm(m1_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
  }
  // AjMODm1 = y;                 // Exponentiation finished:  AjMODm = expMOD = y = a^j

  // -- Compute and display the seeds S(i+j), from the present seed S(i), using the previously calculated value of (a^j)MOD(m):
  //         S(i+j) = [(a**j MOD m)*S(i)] MOD m
  //         S_i = abMODm(m,S_i,AjMODm)
  seed->x = abMODm(m1_RANECU, seed_input, y);     // Using the input seed as the starting seed

  // **** 2nd MLCG (repeating the previous calculation for the 2nd MLCG parameters):
  leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
  y = 1;
  z = a2_RANECU;
  for(;;)
  {
    // (A2) Halve n, and store the integer part and the residue
    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
    {
      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);
      y = abMODm(m2_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
      if (0==leap) break;         // (A4) leap==0? ==> finish
    }
    else           // (leap is even)
    {
      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
    }
    z = abMODm(m2_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
  }
  // AjMODm2 = y;
  seed->y = abMODm(m2_RANECU, seed_input, y);     // Using the input seed as the starting seed
}



/////////////////////////////////////////////////////////////////////
//!  Calculate "(a1*a2) MOD m" with 32-bit integers and avoiding
//!  the possible overflow, using the Russian Peasant approach
//!  modulo m and the approximate factoring method, as described
//!  in:  L'Ecuyer and Cote, ACM Trans. Math. Soft. 17 (1991).
//!
//!  This function has been adapted from "seedsMLCG.f", see: 
//!  Badal and Sempau, Computer Physics Communications 175 (2006)
//!
//!       @param[in] m,a,s  MLCG parameters
//!       @return   (a1*a2) MOD m   
//
//    Input:          0 < a1 < m                                  
//                    0 < a2 < m                                  
//
//    Return value:  (a1*a2) MOD m                                
//
/////////////////////////////////////////////////////////////////////
__device__ __host__    // Function will be callable from host and also from device
inline int abMODm(int m, int a, int s)
{
  // CAUTION: the input parameters are modified in the function but should not be returned to the calling function! (pass by value!)
  int q, k;
  int p = -m;            // p is always negative to avoid overflow when adding

  // ** Apply the Russian peasant method until "a =< 32768":
  while (a>32768)        // We assume '32' bit integers (4 bytes): 2^(('32'-2)/2) = 32768
  {
    if (0!=(a&1))        // Store 's' when 'a' is odd     Equivalent code:   if (1==(a%2))
    {
      p += s;
      if (p>0) p -= m;
    }
    a >>= 1;             // Half a (move bits 1 position right)   Equivalent code: a = a/2;
    s = (s-m) + s;       // Double s (MOD m)
    if (s<0) s += m;     // (s is always positive)
  }

  // ** Employ the approximate factoring method (a is small enough to avoid overflow):
  q = (int) m / a;
  k = (int) s / q;
  s = a*(s-k*q)-k*(m-q*a);
  while (s<0)
    s += m;

  // ** Compute the final result:
  p += s;
  if (p<0) p += m;

  return p;
}



////////////////////////////////////////////////////////////////////////////////
//! Pseudo-random number generator (PRNG) RANECU returning a float value
//! (single precision version).
//!
//!       @param[in,out] seed   PRNG seed (seed kept in the calling function and updated here).
//!       @return   PRN double value in the open interval (0,1)
//!
////////////////////////////////////////////////////////////////////////////////
__device__ 
inline float ranecu(int2* seed)
{
  int i1 = (int)(seed->x/53668);
  seed->x = 40014*(seed->x-i1*53668)-i1*12211;

  int i2 = (int)(seed->y/52774);
  seed->y = 40692*(seed->y-i2*52774)-i2*3791;

  if (seed->x < 0) seed->x += 2147483563;
  if (seed->y < 0) seed->y += 2147483399;

  i2 = seed->x-seed->y;
  if (i2 < 1) i2 += 2147483562;


  return (__int2float_rn(i2)*4.65661305739e-10f);        // 4.65661305739e-10 == 1/2147483563
}


////////////////////////////////////////////////////////////////////////////////
//! Pseudo-random number generator (PRNG) RANECU returning a double value.
////////////////////////////////////////////////////////////////////////////////
__device__ 
inline double ranecu_double(int2* seed)
{
  int i1 = (int)(seed->x/53668);
  seed->x = 40014*(seed->x-i1*53668)-i1*12211;

  int i2 = (int)(seed->y/52774);
  seed->y = 40692*(seed->y-i2*52774)-i2*3791;

  if (seed->x < 0) seed->x += 2147483563;
  if (seed->y < 0) seed->y += 2147483399;

  i2 = seed->x-seed->y;
  if (i2 < 1) i2 += 2147483562;

  return (__int2double_rn(i2)*4.6566130573917692e-10);
}



////////////////////////////////////////////////////////////////////////////////
//! Find the voxel that contains the current position.
//! Report the voxel absolute index and the x,y,z indices.
//! The structure containing the voxel number and size is read from CONSTANT memory.
//!
//!       @param[in] position   Particle position
//!       @return   Returns "absvox", the voxel number where the particle is
//!                 located (negative if position outside the voxel bbox).
//!
////////////////////////////////////////////////////////////////////////////////
__device__
inline int locate_voxel(float3* position)
{

  if ( (position->y < EPS_SOURCE) || (position->y > (voxel_data_CONST.size_bbox.y - EPS_SOURCE)) ||
       (position->x < EPS_SOURCE) || (position->x > (voxel_data_CONST.size_bbox.x - EPS_SOURCE)) ||
       (position->z < EPS_SOURCE) || (position->z > (voxel_data_CONST.size_bbox.z - EPS_SOURCE)) )
  {
    // -- Particle escaped the voxelized geometry (using EPS_SOURCE to avoid numerical precision errors):      
     return -100;
  }
 
  // -- Particle inside the voxelized geometry, find current voxel:
  //    The truncation from float to integer could give troubles for negative coordinates but this will never happen thanks to the IF at the begining of this function.
  //    (no need to use the CUDA function to convert float to integer rounding down (towards minus infinite): __float2int_rd)
  
  register int voxel_coord_x, voxel_coord_y, voxel_coord_z;
  voxel_coord_x = __float2int_rd(position->x * voxel_data_CONST.inv_voxel_size.x);  
  voxel_coord_y = __float2int_rd(position->y * voxel_data_CONST.inv_voxel_size.y);
  voxel_coord_z = __float2int_rd(position->z * voxel_data_CONST.inv_voxel_size.z);

//   // Output the voxel coordinates as short int (2 bytes) instead of int (4 bytes) to save registers; avoid type castings in the calculation of the return value.
//   voxel_coord->x = (short int) voxel_coord_x;
//   voxel_coord->y = (short int) voxel_coord_y;
//   voxel_coord->z = (short int) voxel_coord_z;
  
  return (voxel_coord_x + voxel_coord_y*(voxel_data_CONST.num_voxels.x) + voxel_coord_z*(voxel_data_CONST.num_voxels.x)*(voxel_data_CONST.num_voxels.y));  
}



//////////////////////////////////////////////////////////////////////
//!   Rotates a vector; the rotation is specified by giving
//!   the polar and azimuthal angles in the "self-frame", as
//!   determined by the vector to be rotated.
//!   This function is a literal translation from Fortran to C of
//!   PENELOPE (v. 2006) subroutine "DIRECT".
//!
//!    @param[in,out]  (u,v,w)  input vector (=d) in the lab. frame; returns the rotated vector components in the lab. frame
//!    @param[in]  costh  cos(theta), angle between d before and after turn
//!    @param[in]  phi  azimuthal angle (rad) turned by d in its self-frame
//
//    Output:
//      (u,v,w) -> rotated vector components in the lab. frame
//
//    Comments:
//      -> (u,v,w) should have norm=1 on input; if not, it is
//         renormalized on output, provided norm>0.
//      -> The algorithm is based on considering the turned vector
//         d' expressed in the self-frame S',
//           d' = (sin(th)cos(ph), sin(th)sin(ph), cos(th))
//         and then apply a change of frame from S' to the lab
//         frame. S' is defined as having its z' axis coincident
//         with d, its y' axis perpendicular to z and z' and its
//         x' axis equal to y'*z'. The matrix of the change is then
//                   / uv/rho    -v/rho    u \
//          S ->lab: | vw/rho     u/rho    v |  , rho=(u^2+v^2)^0.5
//                   \ -rho       0        w /
//      -> When rho=0 (w=1 or -1) z and z' are parallel and the y'
//         axis cannot be defined in this way. Instead y' is set to
//         y and therefore either x'=x (if w=1) or x'=-x (w=-1)
//////////////////////////////////////////////////////////////////////
__device__
inline void rotate_double(float3* direction, double costh, double phi)   // !!DeBuG!! The direction vector is single precision but the rotation is performed in doule precision for increased accuracy.
{
  double DXY, NORM, cosphi, sinphi, SDT;
  DXY = direction->x*direction->x + direction->y*direction->y;
  
  sincos(phi, &sinphi,&cosphi);   // Calculate the SIN and COS at the same time.

  // ****  Ensure normalisation
  NORM = DXY + direction->z*direction->z;     // !!DeBuG!! Check if it is really necessary to renormalize in a real simulation!!
  if (fabs(NORM-1.0)>1.0e-14)
  {
    NORM = 1.0/sqrt(NORM);
    direction->x = NORM*direction->x;
    direction->y = NORM*direction->y;
    direction->z = NORM*direction->z;
    DXY = direction->x*direction->x + direction->y*direction->y;
  }
  if (DXY>1.0e-28)
  {
    SDT = sqrt((1.0-costh*costh)/DXY);
    float direction_x_in = direction->x;
    direction->x = direction->x*costh + SDT*(direction_x_in*direction->z*cosphi-direction->y*sinphi);
    direction->y = direction->y*costh+SDT*(direction->y*direction->z*cosphi+direction_x_in*sinphi);
    direction->z = direction->z*costh-DXY*SDT*cosphi;
  }
  else
  {
    SDT = sqrt(1.0-costh*costh);
    direction->y = SDT*sinphi;
    if (direction->z>0.0)
    {
      direction->x = SDT*cosphi;
      direction->z = costh;
    }
    else
    {
      direction->x =-SDT*cosphi;
      direction->z =-costh;
    }
  }
}


//////////////////////////////////////////////////////////////////////


//  ***********************************************************************
//  *   Translation of PENELOPE's "SUBROUTINE GRAa" from FORTRAN77 to C   *
//  ***********************************************************************
//!  Sample a Rayleigh interaction using the sampling algorithm
//!  used in PENELOPE 2006.
//!
//!       @param[in] energy   Particle energy (not modified with Rayleigh)
//!       @param[out] costh_Rayleigh   Cosine of the angular deflection
//!       @param[in] material  Current voxel material
//
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//  C  PENELOPE/PENGEOM (version 2006)                                     C
//  C    Copyright (c) 2001-2006                                           C
//  C    Universitat de Barcelona                                          C
//  C  Permission to use, copy, modify, distribute and sell this software  C
//  C  and its documentation for any purpose is hereby granted without     C
//  C  fee, provided that the above copyright notice appears in all        C
//  C  copies and that both that copyright notice and this permission      C
//  C  notice appear in all supporting documentation. The Universitat de   C
//  C  Barcelona makes no representations about the suitability of this    C
//  C  software for any purpose. It is provided "as is" without express    C
//  C  or implied warranty.                                                C
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//////////////////////////////////////////////////////////////////////
__device__
inline void GRAa(float *energy, double *costh_Rayleigh, int *mat, float *pmax_current, int2 *seed, struct rayleigh_struct* cgra)
{
/*  ****  Energy grid and interpolation constants for the current energy. */
    double  xmax = ((double)*energy) * 8.065535669099010e-5;       // 8.065535669099010e-5 == 2.0*20.6074/510998.918
    double x2max = min_value( (xmax*xmax) , ((double)cgra->xco[(*mat+1)*NP_RAYLEIGH - 1]) );   // Get the last tabulated value of xco for this mat
    
    if (xmax < 0.01)
    {
       do
       {
          *costh_Rayleigh = 1.0 - ranecu_double(seed) * 2.0;
       }
       while ( ranecu_double(seed) > (((*costh_Rayleigh)*(*costh_Rayleigh)+1.0)*0.5) );
       return;
    }

    for(;;)    // (Loop will iterate everytime the sampled value is rejected or above maximum)
    {
      double ru = ranecu_double(seed) * (double)(*pmax_current);    // Pmax for the current energy is entered as a parameter
 
/*  ****  Selection of the interval  (binary search within pre-calculated limits). */
      int itn = (int)(ru * (NP_RAYLEIGH-1));     // 'itn' will never reach the last interval 'NP_RAYLEIGH-1', but this is how RITA is implemented in PENELOPE
      int i__ = (int)cgra->itlco[itn + (*mat)*NP_RAYLEIGH];
      int j   = (int)cgra->ituco[itn + (*mat)*NP_RAYLEIGH];
      
      if ((j - i__) > 1)
      {
        do
        {
          register int k = (i__ + j)>>1;     // >>1 == /2 
          if (ru > cgra->pco[k -1 + (*mat)*NP_RAYLEIGH])
            i__ = k;
          else
            j = k;
        }
        while ((j - i__) > 1);
      }
       
/*  ****  Sampling from the rational inverse cumulative distribution. */
      int index = i__ - 1 + (*mat)*NP_RAYLEIGH;

      double rr = ru - cgra->pco[index];
      double xx;
      if (rr > 1e-16)
      {      
        double d__ = (double)(cgra->pco[index+1] - cgra->pco[index]);
        float aco_index = cgra->aco[index], bco_index = cgra->bco[index], xco_index = cgra->xco[index];   // Avoid multiple accesses to the same global variable

        xx = (double)xco_index + (double)(aco_index + 1.0f + bco_index)* d__* rr / (d__*d__ + (aco_index*d__ + bco_index*rr) * rr) * (double)(cgra->xco[index+1] - xco_index);
        
      }
      else
      {
        xx = cgra->xco[index];
      }
      
      if (xx < x2max)
      {
        // Sampled value below maximum possible value:
        *costh_Rayleigh = 1.0 - 2.0 * xx / x2max;   // !!DeBuG!! costh_Rayleigh in double precision, but not all intermediate steps are!?
        /*  ****  Rejection: */    
        if (ranecu_double(seed) < (((*costh_Rayleigh)*(*costh_Rayleigh) + 1.0)*0.5))
          break;   // Sample value not rejected! break loop and return.
      }
    }
} /* graa */



//////////////////////////////////////////////////////////////////////////


//  ***********************************************************************
//  *   Translation of PENELOPE's "SUBROUTINE GCOa"  from FORTRAN77 to C  *
//  ********************************************************************* *
//!  Random sampling of incoherent (Compton) scattering of photons, using 
//!  the sampling algorithm from PENELOPE 2006:
//!    Relativistic impulse approximation with analytical one-electron Compton profiles

// !!DeBuG!!  In penelope, Doppler broadening is not used for E greater than 5 MeV.
//            We don't use it in GPU to reduce the lines of code and prevent using COMMON/compos/ZT(M)

//!       @param[in,out] energy   incident and final photon energy (eV)
//!       @param[out] costh_Compton   cosine of the polar scattering angle
//!       @param[in] material   Current voxel material
//!       @param[in] seed   RANECU PRNG seed
//
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//  C  PENELOPE/PENGEOM (version 2006)                                     C
//  C    Copyright (c) 2001-2006                                           C
//  C    Universitat de Barcelona                                          C
//  C  Permission to use, copy, modify, distribute and sell this software  C
//  C  and its documentation for any purpose is hereby granted without     C
//  C  fee, provided that the above copyright notice appears in all        C
//  C  copies and that both that copyright notice and this permission      C
//  C  notice appear in all supporting documentation. The Universitat de   C
//  C  Barcelona makes no representations about the suitability of this    C
//  C  software for any purpose. It is provided "as is" without express    C
//  C  or implied warranty.                                                C
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//
//  ************************************************************************

__device__
inline void GCOa(float *energy, double *costh_Compton, int *mat, int2 *seed, struct compton_struct* cgco_SHARED)
{
    float s, a1, s0, af, ek, ek2, ek3, tau, pzomc, taumin;
    float rn[MAX_SHELLS];
    double cdt1;

     // Some variables used in PENELOPE have been eliminated to save register: float aux, taum2, fpzmax, a, a2, ek1 ,rni, xqc, fpz, pac[MAX_SHELLS];

    int i__;
    int my_noscco = cgco_SHARED->noscco[*mat];    // Store the number of oscillators for the input material in a local variable
    
#ifndef USING_CUDA
    static int warning_flag_1 = -1, warning_flag_2 = -1, warning_flag_3 = -1;    // Write warnings for the CPU code, but only once.  !!DeBuG!!
#endif

    ek = *energy * 1.956951306108245e-6f;    // (1.956951306108245e-6 == 1.0/510998.918)
    ek2 = ek * 2.f + 1.f;
    ek3 = ek * ek;
    // ek1 = ek3 - ek2 - 1.;
    taumin = 1.f / ek2;
    // taum2 = taumin * taumin;
    a1 = logf(ek2);
    // a2 = a1 + ek * 2. * (ek + 1.) * taum2;    // a2 was used only once, code moved below


/*  ****  Incoherent scattering function for theta=PI. */

    s0 = 0.0f;
    for (i__ = 0; i__ < my_noscco; i__++)
    {
       register float temp = cgco_SHARED->uico[*mat + i__*MAX_MATERIALS];
       if (temp < *energy)
       {
         register float aux = *energy * (*energy - temp) * 2.f;
         #ifdef USING_CUDA
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) * rsqrtf(aux + aux + temp * temp) * 1.956951306108245e-6f;
             // 1.956951306108245e-6 = 1.0/510998.918f   // Version using the reciprocal of sqrt in CUDA: faster and more accurate!!
         #else
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) / (sqrtf(aux + aux + temp * temp) * 510998.918f);
         #endif
         if (pzomc > 0.0f)
           temp = (0.707106781186545f+pzomc*1.4142135623731f) * (0.707106781186545f+pzomc*1.4142135623731f);
         else
           temp = (0.707106781186545f-pzomc*1.4142135623731f) * (0.707106781186545f-pzomc*1.4142135623731f);

         temp = 0.5f * expf(0.5f - temp);    // Calculate EXP outside the IF to avoid branching

         if (pzomc > 0.0f)
            temp = 1.0f - temp;
                                
         s0 += cgco_SHARED->fco[*mat + i__*MAX_MATERIALS] * temp;
       }
    }
            
/*  ****  Sampling tau. */
    do
    {
      if (ranecu(seed)*/*a2=*/(a1+2.*ek*(ek+1.f)*taumin*taumin) < a1)
      { 
        tau = powf(taumin, ranecu(seed));    // !!DeBuG!!  "powf()" has a big error (7 ULP), the double version has only 2!! 
      }
      else
      {
        tau = sqrtf(1.f + ranecu(seed) * (taumin * taumin - 1.f));
      }

      cdt1 = (double)(1.f-tau) / (((double)tau)*((double)*energy)*1.956951306108245e-6);    // !!DeBuG!! The sampled COS will be double precision, but TAU is not!!!

      if (cdt1 > 2.0) cdt1 = 1.99999999;   // !!DeBuG!! Make sure that precision error in POW, SQRT never gives cdt1>2 ==> costh_Compton<-1
      
  /*  ****  Incoherent scattering function. */
      s = 0.0f;
      for (i__ = 0; i__ < my_noscco; i__++)
      {
        register float temp = cgco_SHARED->uico[*mat + i__*MAX_MATERIALS];
        if (temp < *energy)
        {
          register float aux = (*energy) * (*energy - temp) * ((float)cdt1);

          if ((aux>1.0e-12f)||(temp>1.0e-12f))  // !!DeBuG!! Make sure the SQRT argument is never <0, and that we never get 0/0 -> NaN when aux=temp=0 !!
          {
         #ifdef USING_CUDA
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) * rsqrtf(aux + aux + temp * temp) * 1.956951306108245e-6f;
             // 1.956951306108245e-6 = 1.0/510998.918f   //  Version using the reciprocal of sqrt in CUDA: faster and more accurate!!
         #else
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) / (sqrtf(aux + aux + temp * temp) * 510998.918f);
         #endif

          }
          else
          {
            pzomc = 0.002f;    // !!DeBuG!! Using a rough approximation to a sample value of pzomc found using pure double precision: NOT RIGUROUS! But this code is expected to be used very seldom, only in extreme cases.
            #ifndef USING_CUDA
            if (warning_flag_1<0)
            {
               warning_flag_1 = +1;  // Disable warning, do not show again
               printf("          [... Small numerical precision error detected computing \"pzomc\" in GCOa (this warning will not be repeated).]\n               i__=%d, aux=%.14f, temp=%.14f, pzomc(forced)=%.14f, uico=%.14f, energy=%.7f, cgco_SHARED->fj0=%.14f, mat=%d, cdt1=%.14lf\n", (int)i__, aux, temp, pzomc, cgco_SHARED->uico[*mat+i__*MAX_MATERIALS], *energy, cgco_SHARED->fj0[*mat+i__*MAX_MATERIALS], (int)*mat, cdt1);   // !!DeBuG!!
            }
            #endif                    
          }
          
          temp = pzomc * 1.4142135623731f;
          if (pzomc > 0.0f)
            temp = 0.5f - (temp + 0.70710678118654502f) * (temp + 0.70710678118654502f);   // Calculate exponential argument
          else
            temp = 0.5f - (0.70710678118654502f - temp) * (0.70710678118654502f - temp);

          temp = 0.5f * expf(temp);      // All threads will calculate the expf together
          
          if (pzomc > 0.0f)
            temp = 1.0f - temp;

          s += cgco_SHARED->fco[*mat + i__*MAX_MATERIALS] * temp;
          rn[i__] = temp;
        }        
      }
    } while( (ranecu(seed)*s0) > (s*(1.0f+tau*(/*ek1=*/(ek3 - ek2 - 1.0f)+tau*(ek2+tau*ek3)))/(ek3*tau*(tau*tau+1.0f))) );  //  ****  Rejection function

    *costh_Compton = 1.0 - cdt1;
        
/*  ****  Target electron shell. */
    for (;;)
    {
      register float temp = s*ranecu(seed);
      float pac = 0.0f;

      int ishell = my_noscco - 1;     // First shell will have number 0
      for (i__ = 0; i__ < (my_noscco-1); i__++)    // !!DeBuG!! Iterate to (my_noscco-1) only: the last oscillator is excited in case all other fail (no point in double checking) ??
      {
        pac += cgco_SHARED->fco[*mat + i__*MAX_MATERIALS] * rn[i__];   // !!DeBuG!! pac[] is calculated on the fly to save registers!
        if (pac > temp)       //  pac[] is calculated on the fly to save registers!  
        {
            ishell = i__;
            break;
        }
      }

    /*  ****  Projected momentum of the target electron. */
      temp = ranecu(seed) * rn[ishell];

      if (temp < 0.5f)
      {
        pzomc = (0.70710678118654502f - sqrtf(0.5f - logf(temp + temp))) / (cgco_SHARED->fj0[*mat + ishell * MAX_MATERIALS] * 1.4142135623731f);
      }
      else
      {
        pzomc = (sqrtf(0.5f - logf(2.0f - 2.0f*temp)) - 0.70710678118654502f) / (cgco_SHARED->fj0[*mat + ishell * MAX_MATERIALS] * 1.4142135623731f);
      }
      if (pzomc < -1.0f)
      {
        continue;      // re-start the loop
      }

  /*  ****  F(EP) rejection. */
      temp = tau * (tau - (*costh_Compton) * 2.f) + 1.f;       // this variable was originally called "xqc"
      
        // af = sqrt( max_value(temp,1.0e-30f) ) * (tau * (tau - *costh_Compton) / max_value(temp,1.0e-30f) + 1.f);  //!!DeBuG!! Make sure the SQRT argument is never <0, and that I don't divide by zero!!

      if (temp>1.0e-20f)   // !!DeBuG!! Make sure the SQRT argument is never <0, and that I don't divide by zero!!
      {
        af = sqrtf(temp) * (tau * (tau - ((float)(*costh_Compton))) / temp + 1.f);
      }
      else
      {
        // When using single precision, it is possible (but very uncommon) to get costh_Compton==1 and tau==1; then temp is 0 and 'af' can not be calculated (0/0 -> nan). Analysing the results obtained using double precision, we found that 'af' would be almost 0 in this situation, with an "average" about ~0.002 (this is just a rough estimation, but using af=0 the value would never be rejected below).

        af = 0.00200f;    // !!DeBuG!!
                
        #ifndef USING_CUDA
        if (warning_flag_2<0)
        {
            warning_flag_2 = +1;  // Disable warning, do not show again
            printf("          [... Small numerical precision error detected computing \"af\" in GCOa (this warning will not be repeated)].\n               xqc=%.14f, af(forced)=%.14f, tau=%.14f, costh_Compton=%.14lf\n", temp, af, tau, *costh_Compton);    // !!DeBuG!!
        }
        #endif
      }

      if (af > 0.0f)
      {
        temp = af * 0.2f + 1.f;    // this variable was originally called "fpzmax"
      }
      else
      {
        temp = 1.f - af * 0.2f;
      }
      
      if ( ranecu(seed)*temp < /*fpz =*/(af * max_value( min_value(pzomc,0.2f) , -0.2f ) + 1.f) )
      {
        break;
      }

    }

/*  ****  Energy of the scattered photon. */
    {
      register float t, b1, b2, temp;
      t = pzomc * pzomc;
      b1 = 1.f - t * tau * tau;
      b2 = 1.f - t * tau * ((float)(*costh_Compton));

      temp = sqrtf( fabsf(b2 * b2 - b1 * (1.0f - t)) );
      
          
      if (pzomc < 0.0f)
         temp *= -1.0f;

      // !Error! energy may increase (slightly) due to inacurate calculation!  !!DeBuG!!
      t = (tau / b1) * (b2 + temp);
      if (t > 1.0f)
      {
        #ifndef USING_CUDA

        #endif      
        #ifndef USING_CUDA
        if (warning_flag_3<0)
        {
            warning_flag_3 = +1;  // Disable warning, do not show again
            printf("\n          [... a Compton event tried to increase the x ray energy due to precision error. Keeping initial energy. (This warning will not be repeated.)]\n               scaling=%.14f, costh_Compton=%.14lf\n", t, *costh_Compton);   // !!DeBuG!!
        }
        #endif
        
        t = 1.0f; // !!DeBuG!! Avoid increasing energy by hand!!! not nice!!
      }

      (*energy) *= t;
       // (*energy) *= (tau / b1) * (b2 + temp);    //  Original PENELOPE code
    }
    
}  // [End subroutine GCOa]



////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
//!  Tally the depose deposited inside each material.
//!  This function is called whenever a particle suffers a Compton or photoelectric
//!  interaction. The energy released in each interaction is added and later in the 
//!  report function the total deposited energy is divided by the total mass of the 
//!  material in the voxelized object to get the dose. This naturally accounts for
//!  multiple densities for voxels with the same material (not all voxels have same mass).
//!  Electrons are not transported in MC-GPU and therefore we are approximating
//!  that the dose is equal to the KERMA (energy released by the photons alone).
//!  This approximation is acceptable when there is electronic equilibrium and 
//!  when the range of the secondary electrons is shorter than the organ size. 
//!
//!  The function uses atomic functions for a thread-safe access to the GPU memory.
//!  We can check if this tally was disabled in the input file checking if the array
//!  materials_dose was allocated in the GPU (disabled if pointer = NULL).
//!
//!
//!       @param[in] Edep   Energy deposited in the interaction
//!       @param[in] material   Current material id number
//!       @param[out] materials_dose   ulonglong2 array storing the mateials dose [in eV/g] and dose^2 (ie, uncertainty).
////////////////////////////////////////////////////////////////////////////////
__device__
inline 
void tally_materials_dose(float* Edep, int* material, ulonglong2* materials_dose)
{
      
// !!DeBuG!! The energy can be tallied directly with atomicAdd in global memory or using shared memory first and then global for whole block if too slow. With the initial testing it looks like using global memory is already very fast!

// !!DeBuG!! WARNING: with many histories and few materials the materials_dose integer variables may overflow!! Using double precision floats would be better. Single precision is not good enough because adding small energies to a large counter would give problems.

#ifdef USING_CUDA
  atomicAdd(&materials_dose[*material].x, __float2ull_rn((*Edep)*SCALE_eV) );  // Energy deposited at the material, scaled by the factor SCALE_eV and rounded.
  atomicAdd(&materials_dose[*material].y, __float2ull_rn((*Edep)*(*Edep)) );   // Square of the dose to estimate standard deviation (not using SCALE_eV for std_dev to prevent overflow)
#else
  materials_dose[*material].x += (unsigned long long int)((*Edep)*SCALE_eV + 0.5f);
  materials_dose[*material].y += (unsigned long long int)((*Edep)*(*Edep) + 0.5f);
#endif     
          
  return;
}

//// !!SINOGRAM!!     [March 28, 2018]
////////////////////////////////////////////////////////////////////////////////
//!  Tally a sinogram of the COINCIDENCE photons escaping the voxelized volume.
//!  If no absorption happens, the 2 rays are histogrammed into a sinogram
//!
//!       @param[in] energy    X-ray energy
//!       @param[in] position  Particle position
//!       @param[in] direction Particle direction (cosine vectors)
//!       @param[in] scatter_state  Flag marking primaries, single Compton, single Rayleigh or multiple scattered radiation
//!       @param[in] detector_data  Variables defining the detector geometry, stored in shared memory for fast access.
//!       @param[out] SINOG    Pointer to the global memory array storing the SINOGRAM
////////////////////////////////////////////////////////////////////////////////
__device__
inline void tally_Sinogram(float* energy, float* energyFirst, float3* position, float3* positionFirst, float3* direction, float3* directionFirst, signed char* scatter_state, signed char* scatter_stateFirst, struct detector_struct* detector_data, struct source_struct* source_data, unsigned long long int* acquisition_time_ps_thread, float* travel_distance, float* travel_distanceFirst, int* True_dev, int* Scatter_dev, int* Imagen_T_dev, int* Imagen_SC_dev, int* Energy_Spectrum_dev, int2 *seed, float* E_resol, float* E_low, float* E_high, float* FOVZ, int* NROWS, int* NCRYSTALS, int* NANGLES, int* NRAD, int* NZS, int* NBINS, int* RES, int* NVOXS, int* NE, int* MRD, int* SPAN, int* NSINOS)  {   // !!MCGPU-PET!!    // !!COINCIDENCE!!

      //! Assuming a cylindrical detector with central axis in the Z direction. The cylinder must be larger than the voxel volume (negative distances not computed)       // !!MCGPU-PET!!
      /*
       -OLD HARDCODED DETECTOR. Now data entered in input file!
        float FOVZ = 50.0;   //cm
        float offsetZ =10.0;
        int NCRYSTALS=672;
        int NANGLES=NCRYSTALS/2;
        int NRAD= 336;
        int NZS = 128;
        float DZ = FOVZ/NZS;
        int NBINS = NRAD*NANGLES*NZS;
        const float PIc = 3.1415926536;
        int RES = 256;
        int NVOXS = RES*RES*NZS;   
      */
//----------------------------------------------------------------------------------

  int ibin;
  // -- Move photon to the surface of the cylinder (radius in cylindrical coordinates equals PSF_radius):
  float x0 = position->x - detector_data->PSF_center.x;   // Move to coordinate system with cylinder center at origin
  float y0 = position->y - detector_data->PSF_center.y;
  
  float A = direction->x*direction->x + direction->y*direction->y;
  float B = 2.0f*(x0*direction->x + y0*direction->y);
  float C = x0*x0 + y0*y0 - detector_data->PSF_radius*detector_data->PSF_radius;
    
  float dist;
  if (A==0.0f) {
    dist = 99999999.9f;   // Photon moving along Z axis -> it will intersect the cylinder only at infinite distance
  } else {
    dist = (-B + sqrtf(B*B-4.0f*A*C)) / (2.0f*A);    // There will always be a real, positive dist as long as the cylinder is larger than the voxels (no particle is ever outside the cylinder)
  }

// comment this line and generate new variables for this purpose
//  *travel_distance += dist;    //FEB2022  Variable 'travel_distance' not needed for the Sinogram, only for the PSF.
//  position->x = x0 + dist*direction->x;
//  position->y = y0 + dist*direction->y;
//  position->z = (position->z - detector_data->PSF_center.z) + dist*direction->z;
//  float phi = atan2f(position->y, position->x);
  
  float positionxnew = x0 + dist*direction->x;
  float positionynew = y0 + dist*direction->y;
  float positionznew = (position->z - detector_data->PSF_center.z) + dist*direction->z;
  float phi = atan2f(positionynew, positionxnew);

  // -- Report only particles that intersect the cylinder inside the PSF detector height:
  //if (fabsf(position->z)<0.5f*detector_data->PSF_height) {    
  //new version
    if (fabsf(positionznew)<0.5f*detector_data->PSF_height)  {   

    // Repeat same calculations for second photon:
    x0 = positionFirst->x - detector_data->PSF_center.x;   // Move to coordinate system with cylinder center at origin
    y0 = positionFirst->y - detector_data->PSF_center.y;
  
    A = directionFirst->x*directionFirst->x + directionFirst->y*directionFirst->y;
    B = 2.0f*(x0*directionFirst->x + y0*directionFirst->y);
    C = x0*x0 + y0*y0 - detector_data->PSF_radius*detector_data->PSF_radius;
    
    if (A==0.0f) {
      dist = 99999999.9f;   // Photon moving along Z axis -> it will intersect the cylinder only at infinite distance
    } else {
      dist = (-B + sqrtf(B*B-4.0f*A*C)) / (2.0f*A);    // There will always be a real, positive dist as long as the cylinder is larger than the voxels (no particle is ever outside the cylinder)
    }


//    *travel_distanceFirst += dist;   //FEB2022 
//    positionFirst->x = x0 + dist*directionFirst->x;
//    positionFirst->y = y0 + dist*directionFirst->y;
//    positionFirst->z = (positionFirst->z - detector_data->PSF_center.z) + dist*directionFirst->z;
//    float phi1 = atan2f(positionFirst->y, positionFirst->x);
 
    float positionFirstxnew = x0 + dist*directionFirst->x;
    float positionFirstynew = y0 + dist*directionFirst->y;
    float positionFirstznew = (positionFirst->z - detector_data->PSF_center.z) + dist*directionFirst->z;
    float phi1 = atan2f(positionFirstynew, positionFirstxnew);
 
    // int ivox = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;    

    // -- Report only particles that intersect the cylinder inside the PSF detector height:
    //if (fabsf(positionFirst->z)<0.5f*detector_data->PSF_height)  {    

    if (fabsf(positionFirstznew)<0.5f*detector_data->PSF_height)  {    
/*
    // OLD CODE: now the PSF is reported in a separate routine   
    
//  Safely get two slots in the PSF array in global memory:  
        int index = atomicAdd(index_PSF, 2);
        if (index>2000000000) *index_PSF = 2000000000;    // Prevent overflow of integer counter (max value 2^31-1~2.14e9)      //!!DeBuG!!
        if (index<detector_data->PSF_size) {
        // Store particles in order of arrival (shortest travel first):
        int i0, i1;
        if (*travel_distance<*travel_distanceFirst) {
          i0=index;i1=index+1;
        }else {
          i0=index+1; i1=index;
        }
        
        // Store first particle data in global memory:
        PSF[i0].emission_time_ps = source_data->acquisition_time_ps - *acquisition_time_ps_thread;             // Report emission time starting from time 0 s
        PSF[i0].travel_time_ps   = (*travel_distance)*inv_SPEEDOFLIGHT;                                 // Convert the distance to picoseconds (assuming speed of light in vacuum for every material)
        PSF[i0].emission_absvox  = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;  // Assuming that the grid size is equal to the voxel geomety size!   !!MCGPU-PET!!  
        PSF[i0].energy = *energy;
        PSF[i0].z      = position->z;
        PSF[i0].phi    = phi;
        PSF[i0].vx     = direction->x;
        PSF[i0].vy     = direction->y;
        PSF[i0].vz     = direction->z;
        PSF[i0].index1 = (short int)(*scatter_state);   // Flag for scatter: =0 for non-scattered, =1 for Compton, =2 for Rayleigh, and =3 for multiple scatter.
        PSF[i0].index2 = (short int)0;                  // use not defined yet (decay, prompt...)        
        // Store second particle data in global memory
        PSF[i1].emission_time_ps = source_data->acquisition_time_ps - *acquisition_time_ps_thread;             // Report emission time starting from time 0 s
        PSF[i1].travel_time_ps   = (*travel_distanceFirst)*inv_SPEEDOFLIGHT;                                 // Convert the distance to picoseconds (assuming speed of light in vacuum )
        PSF[i1].emission_absvox  = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;  // Assuming that the grid size is equal to the voxel geomety size!   !!MCGPU-PET!!  
        PSF[i1].energy = *energyFirst;
        PSF[i1].z      = positionFirst->z;
        PSF[i1].phi    = phi1;
        PSF[i1].vx     = directionFirst->x;
        PSF[i1].vy     = directionFirst->y;
        PSF[i1].vz     = directionFirst->z;
        PSF[i1].index1 = (short int)(*scatter_stateFirst);   // Flag for scatter: =0 for non-scattered, =1 for Compton, =2 for Rayleigh, and =3 for multiple scatter.
        PSF[i1].index2 = (short int)0;                       // use not defined yet (decay, prompt...)
*/
//---
int ix1=__float2int_rd((phi/(2.0*PI)+0.5)*(*NCRYSTALS));                  //Indice de posicion en el cilindro
int ix2=__float2int_rd((phi1/(2.0*PI)+0.5)*(*NCRYSTALS)); 
int iz1=__float2int_rd(((positionznew)/(*FOVZ) + 0.5)*(*NROWS));   // Cristal en Z
int iz2=__float2int_rd(((positionFirstznew)/(*FOVZ) + 0.5)*(*NROWS));
//int iz1=__float2int_rd(((position->z)/(*FOVZ) + 0.5)*(*NROWS));   // Cristal en Z
//int iz2=__float2int_rd(((positionFirst->z)/(*FOVZ) + 0.5)*(*NROWS));
//int iz1=__float2int_rd((position->z+offsetZ)*NZS/FOVZ);
//int iz2=__float2int_rd((positionFirst->z+offsetZ)*NZS/FOVZ);

if (iz1<0 || iz2<0 || iz1>=*NROWS || iz2>=*NROWS){ ibin = -1; return;}
if (abs(iz2-iz1)>*MRD) {ibin=-1;return;}
float delta=abs(iz2-iz1);
int incl=__float2int_rd((delta-1+((*SPAN+1)/2))/(*SPAN)); 
int iseg=2*incl;
if (iz2<iz1 && iseg>0) {iseg=iseg-1;} //first negative slopes
int ofseg=iseg*(*NZS);
for (int kka=0; kka<=iseg; kka=kka+1) 
{
float ofk=(kka-2)*0.5;
if (kka>1) {ofseg=ofseg-((*SPAN)+1);}
if (kka>3) {ofseg=ofseg-__float2int_rd(ofk)*2*(*SPAN);}
}
int ofz=0;
if (incl>0) {ofz=ofz+(((*SPAN)+1)/2)+(incl-1)*(*SPAN);} 
int izm =ofseg+(iz1+iz2)-ofz;
//if (threadIdx.x==0 && (ivox%10000)==0) printf("z1= %f  iz1= %d  z2= %f  iz2= %d\n ",position->z,iz1,positionFirst->z,iz2);
int ixtemp = ix2; 
int iztemp = iz2; 
if (ix2<ix1){ix2=ix1;ix1=ixtemp;iz2=iz1; iz1=iztemp;}  // We assume ix2>ix1. So if not, we swap 1 and 2
int ith=((ix1+ix2+(*NANGLES)+1)%(*NCRYSTALS)) / 2;
int ir=abs(ix2-ix1-(*NANGLES));
if ((ix1<ith) || (ix2>=(ith+(*NANGLES)))) {ir = -ir;}

ir = ir+(*NRAD)/2; 
if (izm<0 || izm>=(*NSINOS) || ir<0 || ir>=*NRAD) {
  ibin = -1;
  return;
} else { 
  ibin=izm*(*NANGLES)*(*NRAD) + ith*(*NRAD) + ir; 
}

//--Maximum Ring Difference y GAPS --
//if ((iz1+1)%14==0 ||(iz2+1)%14==0|| abs(iz2-iz1)>*MRD ) {
//  ibin = -1;
//  return;
//} 

//float E_resol = 0.12;  //0.12; // Input Parameter (new version al .in)
float randno1 = ranecu(seed);   
float randno2 = ranecu(seed);   
float gaussian_var1 = sqrtf(-2.0*logf(randno1+1e-8f))*cosf(2.0f*PI*randno2);
float randno3 = ranecu(seed);   
float randno4 = ranecu(seed);   
float gaussian_var2 = sqrtf(-2.0*logf(randno3+1.0e-8f))*cosf(2.0f*PI*randno4);
float Energia1 = (*energy) + ((*E_resol)/2.35f)*(*energy)*gaussian_var1;
float Energia2 = (*energyFirst) + ((*E_resol)/2.35f)*(*energyFirst)*gaussian_var2;

int iE1 = __float2int_rd(Energia1/1000.0f);   //FEB2022  Storing energy spectrum with bin corresponds to 1 keV?
int iE2 = __float2int_rd(Energia2/1000.0f);
if ((iE1<(*NE))&&(iE1>=0)) atomicAdd(&Energy_Spectrum_dev[iE1],1);
if ((iE2<(*NE))&&(iE2>=0)) atomicAdd(&Energy_Spectrum_dev[iE2],1);  


//if (threadIdx.x==0) printf("E1= %f  E2= %f \n",Energia1,Energia2); 
//if (threadIdx.x==0) printf("E_resol= %f  E_low= %f E_high= %f \n",*E_resol,*E_low,*E_high); 

// -- STORING INTO GLOBAL MEMORY --
if ((Energia1)>(*E_low) && (Energia1)<(*E_high) && (Energia2)>(*E_low) && (Energia2)<(*E_high)){

if ((short int)(*scatter_state)==0 && (short int)(*scatter_stateFirst)==0){      // True
  atomicAdd(&True_dev[ibin],1); 
  atomicAdd(&Imagen_T_dev[blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y],1);  // Add one to the emitting voxel 3D counter.     //if (ivox>=0 && ivox<*NVOXS)  atomicAdd(&Imagen_T_dev[ivox],1);
} 

if ((short int)(*scatter_state)>0 || (short int)(*scatter_stateFirst)>0){        // Scatter
  atomicAdd(&Scatter_dev[ibin],1); 
  atomicAdd(&Imagen_SC_dev[blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y],1);        //if (ivox>=0 && ivox<*NVOXS)  atomicAdd(&Imagen_SC_dev[ivox],1);    
}  // True or Scatter

} // energy


//---      

 } // Photon1 within Axial FOV
 } // Photon2 within Axial FOV

} 
